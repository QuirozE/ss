
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void cu_add(int a, int b, int *c) {
    *c = a + b;
}

int main(void) {
    int res;
    int* cuda_res;

    hipMalloc((void**)&cuda_res, sizeof(int));

    cu_add<<<1, 1>>>(10, 7, cuda_res);

    hipMemcpy(&res, cuda_res, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(cuda_res);

    printf("10 + 7 = %d, using a CUDA kernel!!!\n", res);
    return 0;
}
